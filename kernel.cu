#include "hip/hip_runtime.h"

#include <ctime>

#include ""
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include "common.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

//����ʱ�����ȴ洢�����int�ͣ���ȡ��Ӧ�����Ƶĳ���,__clz()ΪCUDA�ڽ�����
//����һ�������Ķ����Ƴ��ȣ�0ʱ����1
__device__ 
int countNum(unsigned int x) {
	//����x��Ӧ�Ķ�����ֵ��λ��,Ϊ0ʱ����1��__clz()Ϊcuda�⺯��
	return x == 0 ? 1 : (32 - __clz(x));
}

//����һ�������Ķ����Ƴ��ȣ�0ʱ����0
__device__ 
int countNum_abc(unsigned int x) {
	return x == 0 ? 0 : (32 - __clz(x));
}

/*
������������Ϊ���뷽����ԭ�ȷ������洢�ַ���
�������ڿ���Ч�ʺ�CUDA�����ƣ���kernel���Ƚ��������Ϊint��
����Host��ͨ��λ����תΪ�ַ����洢
*/
//�����Ķ�Ԫ����,��־λ+codeword 
__device__ 
Code_Len abc(int x, int L, int H) {

	int delta = H - L;
	int range = delta + 1;
	float log2range = log2f(range);
	float ub = ceilf(log2range);
	float lb = floorf(log2range);
	float th = powf(2, ub) - range;
	int residual = x - L;
	Code_Len binCode_S;

	if (residual >= th) {
		residual += th;
		binCode_S.lenth = ub + 1;
	}
	else
		binCode_S.lenth = lb + 1;

	binCode_S.code = residual;
	return binCode_S;
}

//Golomb-Rice���룬���ı���Ϊgrc��־λ+unaryһԪ����+0+binary��Ԫ����
__device__ 
Code_Len grc(int x, int L, int H) {
	int k = 2;  //�̶�k����
	int length_Unary;    //��¼�����ƻ�����̳���
	unsigned int binary;//��Ԫ���벿�֣���Ŀ��ֵ������
	unsigned int u;//��,unary��1�ĸ���
	unsigned int unary;
	Code_Len codewords;

	unsigned int residual;
	if (x < L) {
		residual = L - x - 1; //�����½��Ŀ��ֵ
		codewords.code = 2;
	}
	else {
		residual = x - H - 1; //�����Ͻ��Ŀ��ֵ
		codewords.code = 3;
	}
	//codewords.lenth = 2;

	int len = countNum(residual);
	//��Ϊresidual/2^k ����ȡ����ͨ��λ����>>kλ�����Ƴ��Ķ����Ʋ��ּ�Ϊ������ģ
	//�̾���unary��1��λ��
	if (len <= k) //����������Ƴ���С��k,��λ��Ϊ0������Ϊ0
		length_Unary = 0;
	else          //����k
		length_Unary = len - k;//�̵ĳ��ȣ�һԪ���볤��

	binary = residual & ((1 << k) - 1);//ȡ��������λʱ���Ƴ��Ĳ��֣�������
	u = residual >> k;//�Ƴ��������֣�ȡ���̣���len<=k,��u=0

	unary = (1 << u) - 1;//unary���̵���ֵ��1����������볤�ȼ�d��1��λd����1
	unary = unary << 1 | 1;//��0��ΪһԪ�Ͷ�Ԫ����ķֽ��־

	//codewords.lenth = u + 3 + (len - length_Unary);
	codewords.lenth = u + 3 + k;

	if (codewords.lenth > 16) {
		codewords.code = (codewords.code << 14) | x;	//��8bit��������ֵ�Ķ�����
		codewords.lenth = 16;
		return codewords;
	}

	codewords.code = (codewords.code << ((u + 1) + k)) | (unary << k) | binary;

	//codewords.code = (codewords.code << ((u + 1) + (len - length_Unary))) | (unary << (len - length_Unary)) | binary;
	//codewords.code = ( unary << (2 + len - length_Unary) ) | (binary << 2) | (flag << 1);
	
	return codewords;
}

//�ںˣ��������ȫ��GPU��CUDA������
__global__ 
void Kernel(int rows, int cols, int imageSize, uchar* data, /*unsigned short* code_out, 
	unsigned short* lenth_out, unsigned short* lendata,*/ unsigned char* g_obitres) 
{
	//init shared memory
	__shared__ unsigned char byteStream[8*blockSize];			//byte stream
	__shared__ unsigned char bitStream[blockSize];			//bit stream
	__shared__ unsigned short prefix[blockSize];				//prefix
	__shared__ unsigned char binarylen[blockSize];			//code bit length
	__shared__ unsigned short blockLen;					//total length per block

	int tid = threadIdx.x; //��ǰ���ڵ��̺߳�
	int index = (blockIdx.x * blockDim.x) + threadIdx.x; //���̼߳������
	/***************************************************************************/
	if (index < imageSize) {
		/******************************* Encoding *******************************/
		int N1, N2;
		Code_Len codeword;
		//Ԥ��ģ��
		if (index < 2) {//case1���൱��ԭ���㷨�е�ǰ�������صı���
					//ֱ�Ӽ�¼ԭֵ������Host��תΪ������
			codeword.code = data[index];
			codeword.lenth = 8;
		}
		else if (index < cols) {//case2�����е��������صı���
			N1 = data[index - 2];
			N2 = data[index - 1];
		}
		else if (index % cols == 0) {//case3���������⣬���е��������ص�
			N1 = data[index - cols];
			N2 = data[index - cols + 1];
		}
		else {//case4������
			N1 = data[index - 1];
			N2 = data[index - cols];
		}
		//���뷽ʽѡȡ������
		if (index > 1) {//��ǰ�������ص��߳�
			int L = min(N1, N2);
			int H = max(N1, N2);
			int P = data[index];
			if (P<L || P>H)
				codeword = grc(P, L, H);
			else
				codeword = abc(P, L, H);
		}
		binarylen[tid] = codeword.lenth;
		__syncthreads();
		/******************************** scan to get prefix **********************************/
		int offset = 1;
		bitStream[tid] = 0;
		prefix[tid] = codeword.lenth;		// load binary to l1 cache
		int n = blockSize, tid_offset = tid << 1; 
		for (int d = n >> 1; d > 0; d >>= 1) {
			__syncthreads();
			if (tid < d) {
				int ai = offset * (tid_offset + 1) - 1;
				int bi = offset * (tid_offset + 2) - 1;
				prefix[bi] += prefix[ai];
			}
			offset <<= 1;
		}
		if (tid == 0) prefix[n - 1] = 0;
		for (int d = 1; d < n; d <<= 1) {
			offset >>= 1;
			__syncthreads();
			if (tid < d) {
				int ai = offset * (tid_offset + 1) - 1;
				int bi = offset * (tid_offset + 2) - 1;
				unsigned short t = prefix[ai];
				prefix[ai] = prefix[bi];
				prefix[bi] += t;
			}
		}
		__syncthreads();
		if (tid == 0) {
			blockLen = prefix[n - 1] + binarylen[n - 1];
			//ע����ʵ��bit���Ȳ�ͬ�����һ���ֽ�Ĭ�ϲ���8bit
			unsigned short block_bytenum = blockLen;
			//unsigned short block_bytenum = (blockLen & 7) == 0 ? (blockLen >> 3) : ((blockLen >> 3) + 1);
			bitStream[blockSize-1] = block_bytenum & 0xFF;
			bitStream[blockSize-2] = (block_bytenum >> 8) & 0xFF;
		}
		__syncthreads();
		/******************************** byte stream **********************************/
		unsigned short d = codeword.code;
		for (int i = prefix[tid] + codeword.lenth - 1; i >= prefix[tid]; --i) {
			byteStream[i] = d & 1;
			d >>= 1;
		}
		__syncthreads();
		/******************************** bit stream **********************************/
		if ((tid << 3) < blockLen) {
			int byteIndex = (tid << 3) + 7;
			for (int i = 7; i >= 0 && (byteIndex - i) < blockLen; --i)
				bitStream[tid] |= (byteStream[byteIndex - i] << i);
		}
		g_obitres[index] = bitStream[tid];
	}
}

//��װ�ں˺����ӿڣ���cpp����
void FELICS(int& rows, int& cols, uchar* data, unsigned char* g_obitres, hipStream_t* stream) {
	dim3 block(blockSize, 1);
	dim3 grid(gridSize, 1);
	Kernel << < grid, block, 0 , *stream>> > (I.rows, I.cols, pixelSize, data, g_obitres);
}


